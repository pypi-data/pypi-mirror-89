#include "hip/hip_runtime.h"
#include "lstm.h"
#include "lstm_kernel.h"

void lstm_forward(
    const std::vector<torch::Tensor>& inputs,
    std::vector<torch::Tensor>& outputs,
    const torch::Tensor& one,
    const torch::Tensor& zero) {

    const torch::Tensor& x0 = inputs[0];
    const torch::Tensor& h0 = inputs[1];
    const torch::Tensor& c0 = inputs[2];
    const torch::Tensor& wx = inputs[3];
    const torch::Tensor& wh = inputs[4];
    const torch::Tensor& bias = inputs[5];
    const torch::Tensor& ln_gamma = inputs[6];
    const torch::Tensor& ln_beta = inputs[7];

    torch::Tensor& xbuf = outputs[0];
    torch::Tensor& hbuf = outputs[1];
    torch::Tensor& hn = outputs[2];
    torch::Tensor& cn = outputs[3];
    torch::Tensor& ifog = outputs[4];
    torch::Tensor& ym = outputs[5];
    torch::Tensor& ln_xmu = outputs[6];
    torch::Tensor& ln_xivar = outputs[7];

    const auto seq_len = x0.size(0);
    const auto batch_size = x0.size(1);
    const auto input_size = x0.size(2);
	const auto num_layers = h0.size(0);
    const auto hidden_size = h0.size(2);

    const float* inputptr = x0.data<float>();
    const float* h0ptr = h0.data<float>();
    const float* c0ptr = c0.data<float>();
    const float* wxptr = wx.data<float>();
    const float* whptr = wh.data<float>();
    const float* biasptr = bias.data<float>();
    const float* ln_gammaptr = ln_gamma.data<float>();
    const float* ln_betaptr = ln_beta.data<float>();
    float* xbufptr = xbuf.data<float>();
    float* hbufptr = hbuf.data<float>();
    float* hnptr = hn.data<float>();
    float* cnptr = cn.data<float>();
    float* ifogptr = ifog.data<float>();
    float* outputptr = ym.data<float>();
    float* ln_xmuptr = ln_xmu.data<float>();
    float* ln_xivarptr = ln_xivar.data<float>();
    const float* oneptr = one.data<float>();
    const float* zeroptr = zero.data<float>();

    //double dur;
    //clock_t start,end;
    //start = clock();
	hipblasHandle_t handle;
    checkCublasErr(hipblasCreate(&handle));
    //end = clock();
    //dur = (double)(end - start);
    //printf("hipblasCreate use Time:%f\n",(dur/CLOCKS_PER_SEC));

    // TODO wyr pay attention to wx shape change
    int wxidx[num_layers];
    wxidx[0] = input_size;
    for (int l = 0; l < num_layers - 1; l++) {
        wxidx[l + 1] = hidden_size;
    }
    int wxoffset[num_layers];
    wxoffset[0] = 0;
    for (int l = 0; l < num_layers - 1; l++) {
        wxoffset[l + 1] = wxoffset[l] + wxidx[l] * wxidx[l + 1] * 4;
    }

    for (int l = 0; l < num_layers; l++) {
        //start = clock();
        const float* ln_gamma_x = ln_gammaptr + l * hidden_size * 4 * 2;
        const float* ln_gamma_h = ln_gammaptr + l * hidden_size * 4 * 2 + hidden_size * 4;
        const float* ln_beta_x = ln_betaptr + l * hidden_size * 4 * 2;
        const float* ln_beta_h = ln_betaptr + l * hidden_size * 4 * 2 + hidden_size * 4;
        float* ln_xmu_x = ln_xmuptr + l * seq_len * batch_size * hidden_size * 4 * 2;
        float* ln_xmu_h = ln_xmuptr + l * seq_len * batch_size * hidden_size * 4 * 2 + seq_len * batch_size * hidden_size * 4;
        float* ln_xivar_x = ln_xivarptr + l * seq_len * batch_size * 2;
        float* ln_xivar_h = ln_xivarptr + l * seq_len * batch_size * 2 + seq_len * batch_size;

        //int err = hipDeviceSynchronize();
        //fprintf(stderr, "%s %d, before sync, l: %d, err: %d\n", __FILE__, __LINE__, l, err);
        const float* xdata = (l == 0 ? inputptr : (outputptr + (l - 1) * seq_len * batch_size * hidden_size));
        const float* wxdata = wxptr + wxoffset[l];
        checkCublasErr(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                    hidden_size * 4, seq_len * batch_size, wxidx[l],
                    oneptr, wxdata, hidden_size * 4, xdata, wxidx[l], zeroptr, xbufptr, hidden_size * 4));
        //err = hipDeviceSynchronize();
        //fprintf(stderr, "%s %d, after sync, l: %d, err: %d\n", __FILE__, __LINE__, l, err);

        //end = clock();
        //dur = (double)(end - start);
        //printf("layer gemm use Time:%f\n",(dur/CLOCKS_PER_SEC));    

        //start = clock();

        int blockSize = DEFAULTWARPNUM * WARPSIZE;
        int gridSize = seq_len * batch_size;
        ppl3::cuda::layernorm<float><<<gridSize, blockSize>>>(
                hidden_size * 4, (const float*)xbufptr, ln_gamma_x, ln_beta_x, ln_xmu_x, ln_xivar_x, xbufptr);
        //err = hipDeviceSynchronize();

        //end = clock();
        //dur = (double)(end - start);
        //printf("layer ln use Time:%f\n",(dur/CLOCKS_PER_SEC));    

        //int len = seq_len * batch_size * hidden_size * 4;
        //float hostbuf[len];
        //hipMemcpy(hostbuf, xbufptr, len * sizeof(float), hipMemcpyDeviceToHost);
        //for (int i = 0; i < len; i++) {
        //    fprintf(stderr, "xbufptr[%d]: %lf\n", i, hostbuf[i]);
        //}

        bool hasbias = true;
        const float* biasdata = biasptr + l * (hidden_size * 4);
        const float* whdata = whptr + l * hidden_size * (hidden_size * 4);
        for (int s = 0; s < seq_len; s++) {
            //err = hipDeviceSynchronize();
            //fprintf(stderr, "%s %d, before gemm, l: %d, s: %d, err: %d\n", __FILE__, __LINE__, l, s, err);

            const float* xbufdata = xbufptr + s * batch_size * (hidden_size * 4);
            const float* prehdata = (s == 0 ? (h0ptr + l * batch_size * hidden_size)
                    : (hnptr + (s - 1) * num_layers * batch_size * hidden_size + l * batch_size * hidden_size));
            const float* precdata = (s == 0 ? (c0ptr + l * batch_size * hidden_size)
                    : (cnptr + (s - 1) * num_layers * batch_size * hidden_size + l * batch_size * hidden_size));
            float* hdata = hnptr + s * num_layers * batch_size * hidden_size + l * batch_size * hidden_size;
            float* cdata = cnptr + s * num_layers * batch_size * hidden_size + l * batch_size * hidden_size;
            float* ifogdata = ifogptr + l * seq_len * batch_size * hidden_size * 4 + s * batch_size * hidden_size * 4;
            float* outputdata = outputptr + l * seq_len * batch_size * hidden_size + s * batch_size * hidden_size;
            //start = clock();
            //err = hipDeviceSynchronize();

            //fprintf(stderr, "wh shape: %d %d %d, h shape: %d %d %d, hbuf shape: %d %d\n",
            //        wh.size(0), wh.size(1), wh.size(2), h.size(0), h.size(1), h.size(2), hbuf.size(0), hbuf.size(1));
            checkCublasErr(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                        hidden_size * 4, batch_size, hidden_size,
                        oneptr, whdata, hidden_size * 4, prehdata, hidden_size, zeroptr, hbufptr, hidden_size * 4));

            //err = hipDeviceSynchronize();
            //fprintf(stderr, "%s %d, after gemm, l: %d, s: %d, err: %d\n", __FILE__, __LINE__, l, s, err);
            //err = hipDeviceSynchronize();
            //end = clock();
            //dur = (double)(end - start);
            //printf("seq gemm use time:%f\n",(dur/CLOCKS_PER_SEC));

            //start = clock();

            {
            float* ln_xmu_h_s = ln_xmu_h + s * batch_size * hidden_size * 4;
            float* ln_xivar_h_s = ln_xivar_h + s * batch_size;
            int blockSize = DEFAULTWARPNUM * WARPSIZE;
            int gridSize = batch_size;
            ppl3::cuda::layernorm<float><<<gridSize, blockSize>>>(
                    hidden_size * 4, (const float*)hbufptr, ln_gamma_h, ln_beta_h, ln_xmu_h_s, ln_xivar_h_s, hbufptr);
            }
  
            //err = hipDeviceSynchronize();
            //end = clock();
            //dur = (double)(end - start);
            //printf("seq ln use time:%f\n",(dur/CLOCKS_PER_SEC));

            //start = clock();

            //int len = batch_size * hidden_size * 4;
            //float hostbuf[len];
            //hipMemcpy(hostbuf, hbufptr, len * sizeof(float), hipMemcpyDeviceToHost);
            //for (int i = 0; i < len; i++) {
            //    fprintf(stderr, "hbufptr[%d]: %lf\n", i, hostbuf[i]);
            //}

            {
            dim3 blockSize = {DEFAULTWARPNUM * WARPSIZE, 1, 1};
            dim3 gridSize = {(hidden_size + blockSize.x - 1) / blockSize.x, batch_size, 1};
            ppl3::cuda::activation<float><<<gridSize, blockSize>>>(
                    batch_size, hidden_size, xbufdata , hbufptr, hasbias, biasdata,
                    prehdata, precdata, hdata, cdata, ifogdata, outputdata);
            }

            //err = hipDeviceSynchronize();
            //end = clock();
            //dur = (double)(end - start);
            //printf("seq activation use time:%f\n",(dur/CLOCKS_PER_SEC));
        }
    }
}

void lstm_backward(
    const std::vector<torch::Tensor>& inputs,
    std::vector<torch::Tensor>& outputs,
    const torch::Tensor& one,
    const torch::Tensor& zero) {

    const torch::Tensor& x0 = inputs[0];
    const torch::Tensor& h0 = inputs[1];
    const torch::Tensor& c0 = inputs[2];
    const torch::Tensor& wx = inputs[3];
    const torch::Tensor& wh = inputs[4];
    const torch::Tensor& hn = inputs[5];
    const torch::Tensor& cn = inputs[6];
    const torch::Tensor& ifogbuf = inputs[7];
    const torch::Tensor& ym = inputs[8];
    const torch::Tensor& ln_gamma = inputs[9];
    const torch::Tensor& ln_xmu = inputs[10];
    const torch::Tensor& ln_xivar = inputs[11];

    torch::Tensor& dgatebuf = outputs[0];
    torch::Tensor& xbuf = outputs[1];
    torch::Tensor& hbuf = outputs[2];
    torch::Tensor& dy = outputs[3];
    torch::Tensor& dx = outputs[4];
    torch::Tensor& dh = outputs[5];
    torch::Tensor& dc = outputs[6];
    torch::Tensor& dwx = outputs[7];
    torch::Tensor& dwh = outputs[8];
    torch::Tensor& dbias = outputs[9];
    torch::Tensor& d_ln_gamma = outputs[10];
    torch::Tensor& d_ln_beta = outputs[11];

    const auto seq_len = x0.size(0);
    const auto batch_size = x0.size(1);
    const auto input_size = x0.size(2);
	const auto num_layers = h0.size(0);
    const auto hidden_size = h0.size(2);

    const float* x0ptr = x0.data<float>();
    const float* ifogptr = ifogbuf.data<float>();
    const float* ymptr = ym.data<float>();
    const float* h0ptr = h0.data<float>();
    const float* c0ptr = c0.data<float>();
    const float* hnptr = hn.data<float>();
    const float* cnptr = cn.data<float>();
    const float* wxptr = wx.data<float>();
    const float* whptr = wh.data<float>();
    const float* oneptr = one.data<float>();
    const float* zeroptr = zero.data<float>();
    const float* ln_gammaptr = ln_gamma.data<float>();
    const float* ln_xmuptr = ln_xmu.data<float>();
    const float* ln_xivarptr = ln_xivar.data<float>();
    float* dgatebufptr = dgatebuf.data<float>();
    float* xbufptr = xbuf.data<float>();
    float* hbufptr = hbuf.data<float>();
    float* dyptr = dy.data<float>();
    float* dxptr = dx.data<float>();
    float* dhptr = dh.data<float>();
    float* dcptr = dc.data<float>();
    float* dwxptr = dwx.data<float>();
    float* dwhptr = dwh.data<float>();
    float* dbiasptr = dbias.data<float>();
    float* ln_dgammaptr = d_ln_gamma.data<float>();
    float* ln_dbetaptr = d_ln_beta.data<float>();

	hipblasHandle_t handle;
    checkCublasErr(hipblasCreate(&handle));

    // TODO wyr pay attention to wx shape change
    int wxidx[num_layers + 1];
    wxidx[0] = input_size;
    for (int l = 0; l < num_layers; l++) {
        wxidx[l + 1] = hidden_size;
    }
    int wxoffset[num_layers + 1];
    wxoffset[0] = 0;
    int totalwx = 0;
    for (int l = 0; l < num_layers; l++) {
        totalwx += wxidx[l] * wxidx[l + 1] * 4;
        wxoffset[l + 1] = wxoffset[l] + wxidx[l] * wxidx[l + 1] * 4;
    }

    bool hasbias = true;
    hipMemsetAsync(dxptr, 0, seq_len * batch_size * input_size * sizeof(float));
    hipMemsetAsync(dwxptr, 0, totalwx * sizeof(float));
    hipMemsetAsync(dwhptr, 0, num_layers * hidden_size * hidden_size * 4 * sizeof(float));
    hipMemsetAsync(dbiasptr, 0, num_layers * hidden_size * 4 * sizeof(float));
    hipMemsetAsync(ln_dgammaptr, 0, num_layers * hidden_size * 4 * 2 * sizeof(float));
    hipMemsetAsync(ln_dbetaptr, 0, num_layers * hidden_size * 4 * 2 * sizeof(float));
    for (int l = num_layers - 1; l >= 0; l--) {
        // layernorm
        const float* ln_gamma_x = ln_gammaptr + l * hidden_size * 4 * 2;
        const float* ln_gamma_h = ln_gammaptr + l * hidden_size * 4 * 2 + hidden_size * 4;
        float* ln_dgamma_x = ln_dgammaptr + l * hidden_size * 4 * 2;
        float* ln_dgamma_h = ln_dgammaptr + l * hidden_size * 4 * 2 + hidden_size * 4;
        float* ln_dbeta_x = ln_dbetaptr + l * hidden_size * 4 * 2;
        float* ln_dbeta_h = ln_dbetaptr + l * hidden_size * 4 * 2 + hidden_size * 4;

        // lstm
        const float* wxdata = wxptr + wxoffset[l];
        float* dwxdata = dwxptr + wxoffset[l];
        const float* whdata = whptr + l * hidden_size * hidden_size * 4;
        float* dwhdata = dwhptr + l * hidden_size * hidden_size * 4;
        float* dbiasdata = dbiasptr + l * hidden_size * 4;
        hipMemsetAsync(dhptr, 0, batch_size * hidden_size * sizeof(float));
        hipMemsetAsync(dcptr, 0, batch_size * hidden_size * sizeof(float));
        const float* xlayer = (l == 0 ? x0ptr : (ymptr + (l - 1) * seq_len * batch_size * hidden_size));
        float* dxlayer = (l == 0 ? dxptr : dyptr);
        for (int s = seq_len - 1; s >= 0; s--) {
            const float* cdata = cnptr + s * num_layers * batch_size * hidden_size + l * batch_size * hidden_size;
            const float* prehdata = (s == 0 ? (h0ptr + l * batch_size * hidden_size)
                    : (hnptr + (s - 1) * num_layers * batch_size * hidden_size + l * batch_size * hidden_size));
            const float* precdata = (s == 0 ? (c0ptr + l * batch_size * hidden_size)
                    : (cnptr + (s - 1) * num_layers * batch_size * hidden_size + l * batch_size * hidden_size));
            const float* ifogdata = ifogptr + l * seq_len * batch_size * hidden_size * 4 + s * batch_size * hidden_size * 4;
            const float* dydata = dyptr + s * batch_size * hidden_size;
            const float* xdata = xlayer + s * batch_size * wxidx[l];
            float* dxdata = dxlayer + s * batch_size * wxidx[l];
            {
            dim3 blockSize = {DEFAULTWARPNUM * WARPSIZE, 1, 1};
            dim3 gridSize = {(hidden_size + blockSize.x - 1) / blockSize.x, batch_size, 1};
            ppl3::cuda::activation_backward<float><<<gridSize, blockSize>>>(
                    batch_size, hidden_size, dydata, cdata, precdata, ifogdata,
                    dgatebufptr, dhptr, dcptr, hasbias, dbiasdata);
            }

            // layernorm
            const float* ln_xmu_x = ln_xmuptr + l * seq_len * batch_size * hidden_size * 4 * 2 +
                s * batch_size * hidden_size * 4;
            const float* ln_xmu_h = ln_xmuptr + l * seq_len * batch_size * hidden_size * 4 * 2 +
                seq_len * batch_size * hidden_size * 4 + s * batch_size * hidden_size * 4;
            const float* ln_xivar_x = ln_xivarptr + l * seq_len * batch_size * 2 + s * batch_size;
            const float* ln_xivar_h = ln_xivarptr + l * seq_len * batch_size * 2 + seq_len * batch_size + s * batch_size;
            {
            int blockSize = DEFAULTWARPNUM * WARPSIZE;
            int gridSize = batch_size;
            // xbufptr has seq_len blocks(for fp), bp only use the first block
            ppl3::cuda::layernorm_backward<float><<<gridSize, blockSize>>>(
                    hidden_size * 4, ln_gamma_x, ln_xmu_x, ln_xivar_x, dgatebufptr, ln_dgamma_x, ln_dbeta_x, xbufptr);
            }
            {
            int blockSize = DEFAULTWARPNUM * WARPSIZE;
            int gridSize = batch_size;
            ppl3::cuda::layernorm_backward<float><<<gridSize, blockSize>>>(
                    hidden_size * 4, ln_gamma_h, ln_xmu_h, ln_xivar_h, dgatebufptr, ln_dgamma_h, ln_dbeta_h, hbufptr);
            }
 
            // dwx += torch.matmul(x_t, d_gate)
            checkCublasErr(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        hidden_size * 4, wxidx[l], batch_size,
                        oneptr, xbufptr, hidden_size * 4, xdata, wxidx[l],
                        oneptr, dwxdata, hidden_size * 4));

            // dwh += torch.matmul(h_t, d_gate)
            checkCublasErr(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                        hidden_size * 4, hidden_size, batch_size,
                        oneptr, hbufptr, hidden_size * 4, prehdata, hidden_size,
                        oneptr, dwhdata, hidden_size * 4));

            // dx = torch.matmul(d_gate, wx_t)
            checkCublasErr(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        wxidx[l], batch_size, hidden_size * 4,
                        oneptr, wxdata, hidden_size * 4, xbufptr, hidden_size * 4,
                        zeroptr, dxdata, wxidx[l]));

            // dh = torch.matmul(d_gate, wh_t)
            checkCublasErr(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                        hidden_size, batch_size, hidden_size * 4,
                        oneptr, whdata, hidden_size * 4, hbufptr, hidden_size * 4,
                        zeroptr, dhptr, hidden_size));
        }
    }
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("lstm_forward", &lstm_forward, "lstm forward (CUDA)");
  m.def("lstm_backward", &lstm_backward, "lstm backward (CUDA)");
}
