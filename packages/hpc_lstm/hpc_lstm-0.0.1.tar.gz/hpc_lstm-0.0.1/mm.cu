#include "mm.h"

void mm_forward(
    const torch::Tensor& mata,
    const torch::Tensor& matb,
    torch::Tensor& matc,
    const torch::Tensor& one,
    const torch::Tensor& zero) {
    const auto M = matc.size(0);
    const auto N = matc.size(1);
    const auto K = mata.size(1);

    const float* maptr = mata.data<float>();
    const float* mbptr = matb.data<float>();
    float* mcptr = matc.data<float>();
    const float* oneptr = one.data<float>();
    const float* zeroptr = zero.data<float>();
	hipblasHandle_t handle;
    checkCublasErr(hipblasCreate(&handle));
    checkCublasErr(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                N, M, K, oneptr, mbptr, N, maptr, K, zeroptr, mcptr, N));
}

void mm_backward(
    const torch::Tensor& mata,
    const torch::Tensor& matb,
    const torch::Tensor& gradc,
    torch::Tensor& grada,
    torch::Tensor& gradb,
    const torch::Tensor& one,
    const torch::Tensor& zero) {

    const auto M = gradc.size(0);
    const auto N = gradc.size(1);
    const auto K = mata.size(1);

    const float* maptr = mata.data<float>();
    const float* mbptr = matb.data<float>();
    const float* gcptr = gradc.data<float>();
    float* gaptr = grada.data<float>();
    float* gbptr = gradb.data<float>();
    const float* oneptr = one.data<float>();
    const float* zeroptr = zero.data<float>();
	hipblasHandle_t handle;
    checkCublasErr(hipblasCreate(&handle));
    //checkCublasErr(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
    //            N, M, K, oneptr, mbptr, N, maptr, K, zeroptr, mcptr, N));
    // a: m*k   ga: m*k
    // b: k*n   bt: n*k
    // c: m*n   gc: m*n

    // a * b = c
    // gc * bt = ga, newk = n, newm = m, newn = k, b->bt

    // bt*at=ct
    // a*b=c, c->ct
    // b=bt, d = bt, a * d = c==>dt * at = ct==>b * at = ct

    checkCublasErr(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                K, M, N, oneptr, mbptr, N, gcptr, N, zeroptr, gaptr, K));
    checkCublasErr(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
                N, K, M, oneptr, gcptr, N, maptr, K, zeroptr, gbptr, N));
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("mm_forward", &mm_forward, "mm forward (CUDA)");
  m.def("mm_backward", &mm_backward, "mm backward (CUDA)");
}

