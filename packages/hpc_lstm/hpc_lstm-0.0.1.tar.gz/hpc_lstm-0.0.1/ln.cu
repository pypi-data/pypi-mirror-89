#include "ln.h"
#include "ln_kernel.h"
void ln_forward(
    const torch::Tensor& x,
    const torch::Tensor& gamma,
    const torch::Tensor& beta,
    torch::Tensor& xmu,
    torch::Tensor& xivar,
    torch::Tensor& y) {
    const auto M = x.size(0);
    const auto N = x.size(1);

    const float* xptr = x.data<float>();
    const float* gammaptr = gamma.data<float>();
    const float* betaptr = beta.data<float>();
    float* xmuptr = xmu.data<float>();
    float* xivarptr = xivar.data<float>();
    float* yptr = y.data<float>();

    int blockSize = DEFAULTWARPNUM * WARPSIZE;
    int gridSize = M;
    ppl3::cuda::layernorm<float><<<gridSize, blockSize>>>(N, xptr, gammaptr, betaptr, xmuptr, xivarptr, yptr);
}

void ln_backward(
    const torch::Tensor& gamma,
    const torch::Tensor& xmu,
    const torch::Tensor& xivar,
    const torch::Tensor& grady,
    torch::Tensor& dgamma,
    torch::Tensor& dbeta,
    torch::Tensor& gradx) {
    const auto M = grady.size(0);
    const auto N = grady.size(1);

    const float* gammaptr = gamma.data<float>();
    const float* xmuptr = xmu.data<float>();
    const float* xivarptr = xivar.data<float>();
    const float* gradyptr = grady.data<float>();
    float* dgammaptr = dgamma.data<float>();
    float* dbetaptr = dbeta.data<float>();
    float* gradxptr = gradx.data<float>();

    //int err = hipDeviceSynchronize();
    //int len = M * N;
    //float hostbuf[len];
    //hipMemcpy(hostbuf, xivarptr, len * sizeof(float), hipMemcpyDeviceToHost);
    //for (int i = 0; i < len; i++) {
    //    fprintf(stderr, "xivar data[%d]: %lf\n", i, hostbuf[i]);
    //}

    int blockSize = DEFAULTWARPNUM * WARPSIZE;
    int gridSize = M;
    ppl3::cuda::layernorm_backward<float><<<gridSize, blockSize>>>(N, gammaptr, xmuptr, xivarptr, gradyptr, dgammaptr, dbetaptr, gradxptr);    
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
  m.def("ln_forward", &ln_forward, "ln forward (CUDA)");
  m.def("ln_backward", &ln_backward, "ln backward (CUDA)");
}

